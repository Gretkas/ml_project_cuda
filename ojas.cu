#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <vector>

__global__ void w_ojas(float *x, float *w, const float y, const float learning_rate) {
    size_t i = threadIdx.x;
    float temp = x[i] - y * w[i];
    w[i] = w[i] + learning_rate * y * temp;
};

__device__ float y_ojas(const float *w, const float *x, const int len) {
    float y = 0;
    for (int i = 0; i < len; ++i) {
        y += w[i] * x[i];
    }
    return y;
}

__global__ void y_ojas_par(const float *w, const float *x, float *y) {
    size_t i = threadIdx.x;
    y[i] = w[i] * x[i];
}

__global__ void ojas_rule(float *x, float *w, const float learning_rate, const int num, const int len, const int num_neurons) {
    float y;
    float *x_start;

    const int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_index < num_neurons) {
        
        w = &(w[thread_index * len]);

        for (int i = 0; i < num; ++i) {
            x_start = &(x[i * len]); //Må sende inn riktig deler av x
            y = y_ojas(w, x_start, len);

            w_ojas<<<1, len>>>(x_start, w, y, learning_rate);
            hipDeviceSynchronize();
        }
    }
}

__global__ void ojas_rule_par(float *x, float *w, const float learning_rate, const int num, const int len, const int num_neurons) {
    float *y_arr;
    float y;
    float *x_start;
    y_arr = new float[len];

    const int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_index < num_neurons) {
        
        w = &(w[thread_index * len]);

        for (int i = 0; i < num; ++i) {
            y = 0;
            x_start = &(x[i * len]); //Må sende inn riktig deler av x

            y_ojas_par<<<1, len>>>(w, x_start, y_arr);
            hipDeviceSynchronize();

            for (int i = 0; i < len; ++i) {
                y += y_arr[i];
            }

            w_ojas<<<1, len>>>(x_start, w, y, learning_rate);
            hipDeviceSynchronize();
        }
    }
}

__host__ void run_ojas(float *w, std::vector<float> vec_x, const int num, const int len, const bool par_y, const int num_neurons = 1) {

    float *x = vec_x.data();
    float *d_w, *d_x;
    const float learning_rate = 0.1;
    const size_t x_size = sizeof(*x) * num * len;
    const size_t w_size = sizeof(*w) * len * num_neurons;

    hipMalloc(&d_w, w_size);
    hipMalloc(&d_x, x_size);

    hipMemcpy(d_w, w, w_size, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, x_size, hipMemcpyHostToDevice);

    int num_treads = num_neurons;
    int num_blocks = 1;

    if (num_neurons > 1024) {
        num_treads = 1024;
        num_blocks = (num_neurons + num_treads) / num_treads;
    }

    if (!par_y) {
        ojas_rule<<<num_blocks, num_treads>>>(d_x, d_w, learning_rate, num, len, num_neurons);
    } else {
        ojas_rule_par<<<num_blocks, num_treads>>>(d_x, d_w, learning_rate, num, len, num_neurons);
    }

    hipDeviceSynchronize();
    hipMemcpy(w, d_w, w_size, hipMemcpyDeviceToHost);

    hipFree(d_w);
    hipFree(d_x);
}
