#include "helper.cu"
#include "ojas.cu"
#include <iostream>
#include <vector>
#include <hip/hip_runtime_api.h>

int main() {
    const int num_neurons = 100;                    //ant nevroner som trenes
    const int num_seg = 100;                        //ant segmenter/bilder som algoritmen skal trenes med
    const int len = (5*5); //evt (28*28)            //lengden på et patch/bilde: Patch:(5*5), Bilde:(28*28)
    float *w = generate_w(len * num_neurons);       //skal bare være  (lengde på segment) * (ant nevroner)

    std::cout << "w(0):" << std::endl;
    std::cout << w << std::endl
              << std::endl;

    std::vector<float> x = load_data(num_seg, true); //Set lik false dersom du ønsker å bruke bilder istedenfor bildepatcher/segmenter

    hipProfilerStart();
    run_ojas(w, x, num_seg, len, false, num_neurons); //Bør settes til true dersom man ønsker å regne ut y på en parallellisert måte
    hipProfilerStop();

    std::cout << "w(" << num_seg << "):" << std::endl;
    std::cout << w << std::endl;
}
